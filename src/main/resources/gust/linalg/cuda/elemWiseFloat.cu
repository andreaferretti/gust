
#include <hip/hip_runtime.h>
/*
 * Kernel for calulating the element-wise product of two matrices
 * m, n --> dimensions of matrices A, B, C
 */
extern "C" {
__global__ void hadamard(int m, int n, float *A, int lda, float *B, int ldb, float *C, int ldc)
{
    int i = blockIdx.x + threadIdx.x;
    int j = blockIdx.y + threadIdx.y;

    if (i >= m || j >= n) return;

    C[i + j*ldc] = A[i + j*lda] * B[i + j*ldb];
}
}

/*
 * Matrix sum, parameters as above
 */
extern "C" {
 __global__ void matrix_sum(int m, int n, float *A, int lda, float *B, int ldb, float *C, int ldc)
{
    int i = blockIdx.x + threadIdx.x;
    int j = blockIdx.y + threadIdx.y;

    if (i >= m || j >= n) return;

    C[i + j*ldc] = A[i + j*lda] + B[i + j*ldb];
}
}